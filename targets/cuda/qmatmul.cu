#include "hip/hip_runtime.h"
#include "mico_qnn.h"
#include "profile.h"

#include <hip/hip_runtime.h>

__global__ void q8_matmul_kernel(int32_t *O, const qbyte *x, const qbyte *w, 
                                 size_t batch_size, size_t in_features, size_t out_features) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // batch index
    int col = blockIdx.x * blockDim.x + threadIdx.x; // output feature index

    if (row < batch_size && col < out_features) {
        int32_t acc = 0;
        for (size_t k = 0; k < in_features; k++) {
            acc += x[row * in_features + k] * w[col * in_features + k];
        }
        O[row * out_features + col] = acc;
    }
}

void MiCo_Q8_MatMul(int32_t *O, const Tensor2D_Q8 *x, const Tensor2D_Q8 *w) {
    const size_t batch_size = x->shape[0];
    const size_t in_features = x->shape[1];
    const size_t out_features = w->shape[0];

    dim3 blockSize(16, 16);
    dim3 gridSize((out_features + blockSize.x - 1) / blockSize.x, 
                  (batch_size + blockSize.y - 1) / blockSize.y);
    qbyte *cx, *cw;
    hipMalloc((void**)&cx, batch_size * in_features * sizeof(qbyte));
    hipMalloc((void**)&cw, out_features * in_features * sizeof(qbyte));
    int32_t *cO;
    hipMalloc((void**)&cO, batch_size * out_features * sizeof(int32_t));

    hipMemcpy(cx, x->data, batch_size * in_features * sizeof(qbyte), hipMemcpyHostToDevice);
    hipMemcpy(cw, w->data, out_features * in_features * sizeof(qbyte), hipMemcpyHostToDevice);    
    q8_matmul_kernel<<<gridSize, blockSize>>>(
        cO, cx, cw, batch_size, in_features, out_features);
    hipDeviceSynchronize();
    hipFree(cx);
    hipFree(cw);
    hipMemcpy(O, cO, batch_size * out_features * sizeof(int32_t), hipMemcpyDeviceToHost);
    hipFree(cO);
    return; 
}